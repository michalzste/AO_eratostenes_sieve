
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define BYTE unsigned char


/**
 * @brief Funkcja alokuje ciągły obszar pamięci na karcie graficznej o podanym rozmiarze
 * optymalniej jest zaalokować jeden ciągły obszar niż tysiace mniejszych
 * 
 * @param elements liczba elementów tablicy bitów, którą chcemy zaalokować
 * @param bytesPerBitarray liczba bajtów na tablicę bitów
 * @param bitarrays  liczba tablic bitów, które chcemy zaalokować
 * 
 * @return BYTE* 
 */
inline BYTE *createBitearrays(size_t elements, size_t *bytesPerBitarray, size_t bitarrays) {

  // dodajemy potencjalny bajt, gdy liczba bitów nie jest podzielna przez 8
  *bytesPerBitarray = elements / 8 + ((elements % 8) > 1);  
  BYTE *bitarraysMem;
  hipMalloc(&bitarraysMem, *bytesPerBitarray * bitarrays);
  return bitarraysMem;
}


/**
 * @brief Funkcja uruchamiana przez każdy wątek do szukania liczb pierwszych
 * __global__ oznacza że funkcja zostanie uruchomiona na GPU
 * 
 * @param isPrimeArrays wskaźnik na tablicę bitów zaalokowaną funckją createBitearrays do której będą zapisywane informacje
 * o liczbach pierwszych przez wiele wątków
 * @param isPrimeBytes wielkość tablic bitowych używanych w poszczególnych wątkach (w bajtach)
 * @param defaultPrimeCount makxymalna ilość liczb pierwszych w wątku 
 * (przykładowa tablica ma wielkość 10, parzyste liczby nie mogą być pierwsze więc defaultPrimeCount będzie 5)
 * @param primeCounts liczba znalezionych liczb pierwszych
 * @param chunkCount liczba wątków/wywołań
 * @return __global__ 
 */
__global__ void sieveChunk(BYTE *isPrimeArrays, size_t isPrimeBytes, uint64_t defaultPrimeCount, 
                           uint64_t *primeCounts, uint64_t chunkCount) {

  // index konkretnego wywołania
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (index >= chunkCount) return;
}

int main() {
  printf("%d\n", 0);
  return 0;
}